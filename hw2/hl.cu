#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>

#include <hip/hip_runtime.h>



#define MAX_GRID_DIM 65535
#define SPAWN_LINE_SIZE 10

typedef unsigned char BYTE;


BYTE* world_prev = NULL;
BYTE* world_result = NULL;

size_t world_width, world_height, world_length;

size_t block_dim;
size_t grid_dim;

size_t increments = 1;

#ifdef SHOW_WORLD
int show_world = 1;
#endif

#ifndef SHOW_WORLD
int show_world = 0;
#endif


#ifdef DEBUG
int debug = 1;
#endif

#ifndef DEBUG
int debug = 0;
#endif


// WORLD IS ALL ZEROS
__global__ void init_allzeros(BYTE* wprev, BYTE* wresult, size_t width, size_t height, int _increments){

    size_t jump = gridDim.x * blockDim.x;
    size_t idx;
    for(int i=0;i<_increments;++i){
        idx = blockIdx.x * blockDim.x + threadIdx.x + jump*i;
        if( idx < width * height){
            // check if idx in bounds
            wprev[idx] = 0;
            wresult[idx] = 0;
        }
    }
    

}

// WORLD IS ALL ONES
__global__ void init_allones(BYTE* wprev, BYTE* wresult, size_t width, size_t height, int _increments){

    size_t jump = gridDim.x * blockDim.x;
    size_t idx;
    for(int i=0;i<_increments;++i){
        idx = blockIdx.x * blockDim.x + threadIdx.x + jump*i;
        if( idx < width * height){
            // check if idx in bounds
            wprev[idx] = 1;
            wresult[idx] = 0;
        }
    }

    //__syncthreads();
}

// STREAK OF 10 ONES IN ABOUT THE MIDDLE OF THE WORLD
__global__ void pattern_two(BYTE* wprev, BYTE* wresult, size_t  width, size_t height, int cell_count){

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t centerpos = width*height/2;

    if(idx < cell_count){
        idx += centerpos - cell_count / 2;
        wprev[idx] = 1;
    }
}

// ONES AT THE CORNERS OF THE WORLD
__global__ void pattern_three(BYTE* wprev, BYTE* wresult, size_t width, size_t height, int _increments){
    // all we need is 4 threads...?
}


// SPINNER PATTERN AT CORNERS OF WORLD
__global__ void pattern_four(BYTE* wprev, BYTE* wresult, size_t width, size_t height, int _increments){
    
}


// REPLICATOR PATTERN STARTING IN THE MIDDLE
__global__ void pattern_five(BYTE* wprev, BYTE* wresult, size_t width, size_t height){

    // only need 6 threads
    size_t x = width/2;
    size_t y = height/2;

    // this is dumb but im lazy and its just as fast

    switch(threadIdx.x)
    {
        case 0:
            wprev[x + y*width + 1] = 1;
        break;
        case 1:
            wprev[x + y*width + 2] = 1;
        break;
        case 2:
            wprev[x + y*width + 3] = 1;
        break;
        case 3:
            wprev[x+(y+1)*width] = 1;
        break;
        case 4:
            wprev[x+(y+2)*width] = 1;
        break;
        case 5:
            wprev[x+(y+3)*width] = 1;
        break;
    }
    
}

// core game functions
// game_initMaster: allocates memory and initializes worlds based on pattern
// game_kernelLaunch: driver function to start game
// __global__ game_kernelRun: launches CUDA kernel and runs one iteration of world
// game_swap: swaps prev and next world pointers (device memory)

// cant really increment within thread... since we need synchronization


void game_initMaster(int pattern){
    // allocates device memory for world, calls world init functions

    if(debug){
        printf("creating worlds of size %d x %d\n", world_width, world_height);
    }

    hipError_t out = hipMalloc((void**)&world_prev, world_length);
    if(out){
        fprintf(stderr, "hipMalloc failed, error code %d\n", out);
    }

    out = hipMalloc((void**)&world_result, world_length);
    if(out){
        fprintf(stderr, "hipMalloc failed, error code %d\n", out);
    }

    switch(pattern){
        case 0:
            init_allzeros<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);
        break;

        case 1:
            printf("calling init_allones\n");
            init_allones<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);
        break;
        
        case 2:
            init_allzeros<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);
            hipDeviceSynchronize();
            // dont need to launch shit ton of threads for spawning the line, only 10
            pattern_two<<<1, SPAWN_LINE_SIZE>>>(world_prev, world_result, world_width, world_height, SPAWN_LINE_SIZE);
        break;
            
        case 3:
            init_allzeros<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);

        break;

        case 4:
            init_allzeros<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);

        break;

        case 5:
            init_allzeros<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);
            // 6 threads for replicator pattern
            pattern_five<<<1, 6>>>(world_prev, world_result, world_width, world_height);
        break;
    }

}

__device__ unsigned int check_cell(BYTE* world, int x, int y, size_t width, size_t height){
    if(x < 0 || x >= width || y < 0 || y >= height){
        return 0;
    }
    else{
        size_t idx = y * width + x;
        return world[idx];
    }
}

__global__ void kernel_run(BYTE* wprev, BYTE* wresult, size_t width, size_t height, int _increments){

    size_t jump = gridDim.x * blockDim.x;

    for(int i=0;i<_increments;++i){
        // compute where in array we need to update
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x + jump * i;
        if(idx < width*height){
            size_t x = idx % width;
            size_t y = idx / width;
            int alive = check_cell(wprev, x-1, y, width, height) +
                        check_cell(wprev, x+1, y-1, width, height) +
                        check_cell(wprev, x, y-1, width, height) +
                        check_cell(wprev, x-1, y-1, width, height) +
                        check_cell(wprev, x+1, y, width, height) +
                        check_cell(wprev, x-1, y+1, width, height) +
                        check_cell(wprev, x, y+1, width, height) +
                        check_cell(wprev, x+1, y+1, width, height);
            wresult[idx] = (alive == 3) || (alive == 6 && !wprev[idx]) || (alive == 2 && wprev[idx]) ? 1 : 0;
        }
    }
}

static inline void swap_worlds(BYTE **A, BYTE **B){
    BYTE* temp = *A;
    *A = *B;
    *B = temp;
}

void print_world(BYTE* world, size_t dim){

    
    if(dim > 64){
        fprintf(stderr, "cannot print world larger than dim 64\n");
    }
    else{
        // have to copy device mem back to host mem
        BYTE* hostworld = (BYTE*)calloc(dim*dim, sizeof(BYTE));
        hipError_t out = hipMemcpy(hostworld, world, dim*dim, hipMemcpyDeviceToHost);
        if(out){
            fprintf(stderr, "hipMemcpy failed with %d\n", out);
        }
        for(int i=0;i<dim;++i){
            for(int j=0;j<dim;++j){
                int idx = i*dim + j;
                if(hostworld[idx]){
                    printf("#");
                }else{
                    printf("-");
                }
            }
            printf("\n");
        }
        free(hostworld);
    }

}

void kernel_launch(int iterations){
    // assume world is already constructed

    for(int i=0;i<iterations;++i){
        if(debug){
            printf("Running iteration %d\n", i);
        }
        kernel_run<<<grid_dim, block_dim>>>(world_prev, world_result, world_width, world_height, increments);

        hipError_t out = hipDeviceSynchronize();
        if(out){
           fprintf(stderr, "hipDeviceSynchronize failed with code %d\n", out);
        }

        if(show_world){
            print_world(world_result, world_height);
        }

        swap_worlds(&world_prev, &world_result);
        
    }
    
    printf("-- RUNNING %d KERNEL ITERATIONS WITH: --\n", iterations);
    printf("World dim: %d x %d\n", world_width, world_height);
    printf("World length: %zu\n", world_length);
    printf("Increments: %d\n\n", increments);
    printf("Blocks: %d\n", grid_dim);
    printf("Threads per block: %d\n",  block_dim);
    printf("Cells reachable: %d\n", grid_dim * block_dim);
    printf("Estimated worlds memory usage: %d MiB\n\n", world_length>>(19));
}

inline static void game_free(){
    hipError_t out = hipFree(world_prev);
    if(out){
        fprintf(stderr, "hipFree failed with code %d\n", out);
    }

    out = hipFree(world_result);
    if(out){
        fprintf(stderr, "hipFree failed with code %d\n", out);
    }
}

size_t count_total_alive_cells(BYTE* world){
    printf("counting world cells\n");
    size_t total = 0;
    for(size_t i=0;i<world_length;++i){
        if(world[i]){
            total++;
        }
    }
    return total;
}



int main(int argc, char** argv){


    if(argc!=5){
        printf("usage: ./hl.out <pattern> <world_dim> <iter> <block_dim>\n");
        exit(EXIT_FAILURE);
    }

    int pattern = atoi(argv[1]);
    int world_dim = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    block_dim = atoi(argv[4]);

    world_width = world_dim;
    world_height = world_dim;
    world_length = world_width * world_height;


    // we need to figure out exactly how many blocks we need... starting with 1
    // increment blocks to 65535 - then start increasing repetitions

    double blocks_req = ((double)world_length) / ((double)block_dim);

    if( blocks_req < MAX_GRID_DIM ){
        grid_dim = ceil(blocks_req);
    }
    else{
        grid_dim = 65535;
        // calculate increments
        size_t reachable = grid_dim * block_dim;
        increments = ceil( ((double)world_length) / ((double)reachable));
    }

    game_initMaster(pattern);

    hipError_t out;
    BYTE* hostworld = NULL;

    if(debug){

        hostworld = (BYTE*)calloc(world_length, sizeof(BYTE));
        if(!hostworld){
            fprintf(stderr, "hostworld alloc failed\n");
        }

        out = hipMemcpy(hostworld, world_prev, world_length, hipMemcpyDeviceToHost);
        if(out){
            fprintf(stderr, "hipMemcpy failed with code %d\n", out);
        }

        printf("world starting with %zu alive cells\n", count_total_alive_cells(hostworld));
    }
    

    out = hipDeviceSynchronize();
    if(out){
        fprintf(stderr, "cudaDevicSynchronize failed with code %d\n", out);
    }

    kernel_launch(iterations);

    if(debug){
        out = hipMemcpy(hostworld, world_prev, world_length, hipMemcpyDeviceToHost);
        if(out){
            fprintf(stderr, "hipMemcpy failed with code %d\n", out);
        }

        printf("world ended with %zu alive cells\n", count_total_alive_cells(hostworld));
        free(hostworld);
    }

    game_free();

    return EXIT_SUCCESS;
}
